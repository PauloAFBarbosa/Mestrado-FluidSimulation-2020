#include "hip/hip_runtime.h"
#include "mySort.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>




void mysort(int ** index1, float4 ** values1, int** index2, float4 ** values2,int particles){
	
	thrust::device_ptr<int> i1buff = thrust::device_pointer_cast(*(index1));
	thrust::device_ptr<float4> v1buff = thrust::device_pointer_cast(*(values1));
	thrust::device_ptr<int> i2buff = thrust::device_pointer_cast(*(index2));
	thrust::device_ptr<float4> v2buff = thrust::device_pointer_cast(*(values2));


	thrust::stable_sort_by_key(i1buff, i1buff + particles,v1buff);
	//hipDeviceSynchronize();
	thrust::stable_sort_by_key(i2buff, i2buff + particles, v2buff);

	//hipDeviceSynchronize();
}

