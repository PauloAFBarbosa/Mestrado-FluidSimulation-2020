#include "hip/hip_runtime.h"
#include "mySort.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "cutil_math.h"
//#include <math_functions.h>



void mysort(int * index1, float4 * values1, int* index2, float4 * values2,int particles){
	
	
	
	thrust::device_ptr<int> i1buff = thrust::device_pointer_cast((index1));
	thrust::device_ptr<float4> v1buff = thrust::device_pointer_cast((values1));
	thrust::device_ptr<int> i2buff = thrust::device_pointer_cast((index2));
	thrust::device_ptr<float4> v2buff = thrust::device_pointer_cast((values2));

	
	thrust::sort_by_key(i1buff, i1buff + particles, v1buff);
	thrust::sort_by_key(i2buff, i2buff + particles, v2buff);
	
}

//functions for density Pressure
//Morton code --------------------------------------------


__device__
unsigned int part1by2(unsigned int n) {
    n &= 0x000003ff;
    n = (n ^ (n << 16)) & 0xff0000ff;
    n = (n ^ (n << 8)) & 0x0300f00f;
    n = (n ^ (n << 4)) & 0x030c30c3;
    n = (n ^ (n << 2)) & 0x09249249;
    return n;
}

__device__
unsigned int unpart1by2(unsigned int n) {
    n &= 0x09249249;
    n = (n ^ (n >> 2)) & 0x030c30c3;
    n = (n ^ (n >> 4)) & 0x0300f00f;
    n = (n ^ (n >> 8)) & 0xff0000ff;
    n = (n ^ (n >> 16)) & 0x000003ff;
    return n;
}

__device__
unsigned int interleave3(unsigned int x, unsigned int y, unsigned int z) {
    return part1by2(x) | (part1by2(y) << 1) | (part1by2(z) << 2);
}

__device__
void deinterleave3(unsigned int n, unsigned int x, unsigned int y, unsigned int z) {
    x = unpart1by2(n);
    y = unpart1by2(n >> 1);
    z = unpart1by2(n >> 2);
}


__device__
bool contains(unsigned int arr[27], int size, unsigned int member) {
    bool ret = false;

    for (int i = 0; i < size; i++)
    {
        if (arr[i] == member)
            ret = true;
    }
    return ret;
}
__device__
int getAdj(float4 pos, float H,unsigned int ret[27]) {

    int retSize = 0;
    unsigned int offset = 43;
    unsigned int morton_x = unsigned int((pos.x / H) + offset);
    unsigned int morton_y = unsigned int((pos.y / H) + offset);
    unsigned int morton_z = unsigned int((pos.z / H) + offset);

    unsigned int morton_cell;

    if (morton_x > 0 && morton_y > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y - 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x, morton_y - 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_z > 0) {
        morton_cell = interleave3(morton_x, morton_y, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0 && morton_y > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y - 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y > 0) {
        morton_cell = interleave3(morton_x, morton_y - 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    morton_cell = interleave3(morton_x, morton_y, morton_z);
    ret[retSize] = morton_cell;
    retSize++;

    //1864184 � o numero maximo que o morton code pode devolver num cubo de -2 a 2 
    if (morton_x < 1864184 && morton_y > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y - 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x < 1864184 && morton_y > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y - 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x > 0 && morton_y < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y + 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_y < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x, morton_y + 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_y < 1864184 && morton_x > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y + 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y < 1864184) {
        morton_cell = interleave3(morton_x, morton_y + 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y + 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y + 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x > 0 && morton_y > 0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x - 1, morton_y - 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y > 0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x, morton_y - 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x - 1, morton_y, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_z < 1864184) {
        morton_cell = interleave3(morton_x, morton_y, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y >0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y - 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x > 0 && morton_y < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x - 1, morton_y + 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_y < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x, morton_y + 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y + 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    return retSize;

}

__device__
float useDefaultKernel(float4 distVector, float supportRadius) {
    
    float dist = length(distVector);
    
    if (dist > supportRadius) {

        return 0.0;
    }
    else {
        //printf("Vizinho e vai devolver -> %f \n", (315 / (64 * 3.141592653589793 * pow(supportRadius, 9.0f))) * pow(supportRadius * supportRadius - dist * dist, 3.0f));
        return (315 / (64 * 3.141592653589793 * pow(supportRadius, 9.0f))) * pow(supportRadius * supportRadius - dist * dist, 3.0f);
    }
}
__device__
unsigned int hashFunction(float4 pos, double H, int size) {
    
    int p1 = 2693;
    int p2 = 3163;
    int p3 = 4091;

    int part1 = (int((pos.x / H)) * p1);
    int part2 = (int((pos.y / H)) * p2);
    int part3 = (int((pos.z / H)) * p3);

    unsigned int ret = unsigned int((part1 ^ part2 ^ part3) % size);

    return ret;
}
//----------------


__global__
void count(int * indexes , int* CellStart, int* CellEnd)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	int indexCount = indexes[i];
	atomicAdd(CellStart + indexCount, 1);

	atomicAdd(CellEnd + indexCount, 1);
	
}

void kernelWraper(int * dptrssboIndex, int* dptrssboCellStart, int* dptrssboCellEnd) {

	count<<<6750, 32 >>>(dptrssboIndex, dptrssboCellStart, dptrssboCellEnd);
	
	

	thrust::device_ptr<int> cellstartThrust = thrust::device_pointer_cast((dptrssboCellStart));
	thrust::exclusive_scan(cellstartThrust, cellstartThrust + 2000000, cellstartThrust);
	
}



__global__
void densityPressureKernel(float4 * dptrssboPosition,int* dptrssboIndex, int* dptrssboCellStart, int* dptrssboCellEnd, float* dptrssboDensity, float* dptrssboPressure, int* dptrssboAdj)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    int conta = 0;

    unsigned int ret[27];
    int retSize = 0;
    float H = 0.0457;
    unsigned int offset = 43;
    unsigned int morton_x = unsigned int((dptrssboPosition[index].x / H) + offset);
    unsigned int morton_y = unsigned int((dptrssboPosition[index].y / H) + offset);
    unsigned int morton_z = unsigned int((dptrssboPosition[index].z / H) + offset);
    retSize = getAdj(dptrssboPosition[index], H, ret);

    //printf("Chegou aqui 1\n");
    
    // compute density
    float sum = 1;

    //mudar isto para um var local
    dptrssboAdj[index * 500] = 0;
    //printf("Chegou aqui 2\n");
    int vizinhos = 0;

    int naovizinhos = 0;

    for (int j = 0; j < retSize; j++)
    {
        //uint bucket = uint(adjMat[(28*b)+(j+1)]) ;
        unsigned int cell = ret[j];

        int from = dptrssboCellStart[cell];
        int to = from + dptrssboCellEnd[cell];
        //printf("Chegou aqui 3\n");
        for (int i = from; i < to; i++)
        {


            float4 arg;
            arg = dptrssboPosition[index] - dptrssboPosition[i];

            float ker_res = useDefaultKernel(arg, H);

            sum += 0.02 * ker_res;
            //printf("Chegou aqui 4\n");

            if (ker_res != 0) {

                conta++;
                int vizinhos = dptrssboAdj[index * 500];
                //Aqui � MAXADJ porque no maximo vai dar para guardar 99 vizinhos mais 1 posi��o para dizer quantos vizinhos se guardou
                dptrssboAdj[index * 500 + vizinhos + 1] = i;
                dptrssboAdj[index * 500]++;
                vizinhos++;
                //printf("Chegou aqui 5\n");
            }
            else
                naovizinhos++;


        }
    }

    //printf("Chegou aqui 6 vizinhos %d naovizinhos %d\n",vizinhos,naovizinhos);
    dptrssboDensity[index] = sum;

    //printf("Chegou aqui 7\n");
    // compute pressure
    dptrssboPressure[index] = 3.0 * (sum - 998.29);
}

__device__
float useDefaultKernel_laplacian(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius)
        return 0.0f;
    else
        return -(945 / (32 * 3.141592653589793 * pow(supportRadius, 9.0f))) * (supportRadius * supportRadius - dist * dist) * (3 * supportRadius * supportRadius - 7 * dist * dist);
}
__device__
float useViscosityKernel_laplacian(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius)
        return 0.0f;
    else
        return (45 / (3.141592653589793 * pow(supportRadius, 6.0f))) * (supportRadius - dist);
}
__device__
float4 useDefaultKernel_gradient(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius) {
        return make_float4(0);
    }
    else {
        return -(distVector * (945 / (32 * 3.141592653589793 * pow(supportRadius, 9.0f))) * pow(supportRadius * supportRadius - dist * dist, 2.0f));
    }
}



__device__
float4 usePressureKernel_gradient(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius) {

        return make_float4(0);

    }
    else
    {

        float4 normalized;
        normalized = normalize(distVector);

        return -(normalized * (45 / (3.141592653589793 * pow(supportRadius, 6.0f))) * pow(supportRadius - dist, 2.0f));

    }
}

__global__
void ForceKernel(float4* dptrssboPosition, float4* dptrssboVelocity, float4* dptrssboForce, float* dptrssboDensity, float* dptrssboPressure, float4* dptrssboGravity, float4* dptrssboSurfaceNormal, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, int* dptrssboAdj)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ret[27];
    int retSize = 0;
    float H = 0.0457;

    retSize = getAdj(dptrssboPosition[index], H, ret);

    float4 sumViscosity = make_float4(0);

    float4 sumForce= make_float4(0);
    

    float4 sumSurfaceNormal= make_float4(0);
    
    float sum = 0;
    float MASS = 0.02;

    int count;
    count = 0;

    int vizinhos = dptrssboAdj[index * 500];

    for (int i = 0; i < vizinhos; i++) {
        //Viscosity

        if (dptrssboPosition[index].x == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].x && dptrssboPosition[index].y == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].y && dptrssboPosition[index].z == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].z)
            continue;
        float4 arg;
        arg = dptrssboPosition[index] - dptrssboPosition[dptrssboAdj[index * 500 + i + 1]];

        sumViscosity += useViscosityKernel_laplacian(arg, H) * (dptrssboVelocity[dptrssboAdj[index * 500 + i + 1]] - dptrssboVelocity[index]) * (MASS / dptrssboDensity[dptrssboAdj[index * 500 + i + 1]]);

        //Force
  
        arg = dptrssboPosition[index] - dptrssboPosition[dptrssboAdj[index * 500 + i + 1]];

        
        //printf("ret Kernel %f %f %f \n", usePressureKernel_gradient(arg, H).x, usePressureKernel_gradient(arg, H).y, usePressureKernel_gradient(arg, H).z);
        sumForce += usePressureKernel_gradient(arg, H) * (dptrssboPressure[index] / (dptrssboDensity[index] * dptrssboDensity[index]) + dptrssboPressure[dptrssboAdj[index * 500 + i + 1]] / (dptrssboDensity[dptrssboAdj[index * 500 + i + 1]] * dptrssboDensity[dptrssboAdj[index * 500 + i + 1]])) * MASS;

        //SurfaceNormal
        arg = dptrssboPosition[index] - dptrssboPosition[dptrssboAdj[index * 500 + i + 1]];
        
        sumSurfaceNormal += useDefaultKernel_gradient(arg, H) * (MASS / dptrssboDensity[dptrssboAdj[index * 500 + i + 1]]);
    }

    dptrssboViscosity[index] = sumViscosity * 3.5;

    dptrssboForce[index] = -(sumForce * dptrssboDensity[index]);
    //debug[index] = vec4(density[index]);

    //tempPosition[index]=vec4(sumForce[0],sumForce[1],sumForce[2 ],0);

    dptrssboSurfaceNormal[index] = sumSurfaceNormal;

    dptrssboGravity[index] = make_float4(0.0,-9.8,0.0,0.0) * dptrssboDensity[index];



    if (length(dptrssboSurfaceNormal[index]) >= 7.065) {

        for (int i = 0; i < vizinhos; i++) {
            if (dptrssboPosition[index].x == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].x && dptrssboPosition[index].y == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].y && dptrssboPosition[index].z == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].z)
                continue;
            float4 arg;
            arg = dptrssboPosition[index] - dptrssboPosition[dptrssboAdj[index * 500 + i + 1]];
            
            
            sum += (MASS / dptrssboDensity[dptrssboAdj[index * 500 + i + 1]]) * useDefaultKernel_laplacian(arg, H);
        }




        float4 surfaceNormalNormalized;

        surfaceNormalNormalized = normalize(dptrssboSurfaceNormal[index]);

        dptrssboSurfaceTension[index] = -(surfaceNormalNormalized * 0.0728 * sum);

    }
    else {
        dptrssboSurfaceTension[index] =make_float4(0);
    }
}

void cudaDensityPressure(float4 * dptrssboPosition,int * dptrssboIndex,int * dptrssboCellStart,int * dptrssboCellEnd,float * dptrssboDensity, float * dptrssboPressure, int * dptrssboAdj) {
	densityPressureKernel << <6750, 32 >> > (dptrssboPosition,dptrssboIndex, dptrssboCellStart, dptrssboCellEnd, dptrssboDensity, dptrssboPressure, dptrssboAdj);
}

void cudaForce(float4* dptrssboPosition, float4* dptrssboVelocity, float4* dptrssboForce, float* dptrssboDensity, float* dptrssboPressure, float4* dptrssboGravity, float4* dptrssboSurfaceNormal, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, int* dptrssboAdj) {
    ForceKernel << <6750, 32 >> > (dptrssboPosition, dptrssboVelocity, dptrssboForce, dptrssboDensity, dptrssboPressure, dptrssboGravity, dptrssboSurfaceNormal, dptrssboSurfaceTension, dptrssboViscosity, dptrssboAdj);
}
