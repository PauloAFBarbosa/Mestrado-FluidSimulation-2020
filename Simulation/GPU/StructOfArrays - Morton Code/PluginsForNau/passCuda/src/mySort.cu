#include "mySort.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>




void mysort(int ** dptrssbo){
	
	thrust::device_ptr<int> buff = thrust::device_pointer_cast(*(dptrssbo));
	thrust::stable_sort(buff, buff + 20000000);
}

