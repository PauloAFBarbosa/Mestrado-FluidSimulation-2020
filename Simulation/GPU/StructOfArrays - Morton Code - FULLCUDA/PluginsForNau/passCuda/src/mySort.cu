#include "hip/hip_runtime.h"
﻿#include "mySort.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "cutil_math.h"
//#include <math_functions.h>



void mysort(int * index1, float4 * values1, int* index2, float4 * values2,int particles){
	
	
	
	thrust::device_ptr<int> i1buff = thrust::device_pointer_cast((index1));
	thrust::device_ptr<float4> v1buff = thrust::device_pointer_cast((values1));
	thrust::device_ptr<int> i2buff = thrust::device_pointer_cast((index2));
	thrust::device_ptr<float4> v2buff = thrust::device_pointer_cast((values2));

	
	thrust::sort_by_key(i1buff, i1buff + particles, v1buff);
	thrust::sort_by_key(i2buff, i2buff + particles, v2buff);
	
}

//functions for density Pressure
//Morton code --------------------------------------------


__device__
unsigned int part1by2(unsigned int n) {
    n &= 0x000003ff;
    n = (n ^ (n << 16)) & 0xff0000ff;
    n = (n ^ (n << 8)) & 0x0300f00f;
    n = (n ^ (n << 4)) & 0x030c30c3;
    n = (n ^ (n << 2)) & 0x09249249;
    return n;
}

__device__
unsigned int unpart1by2(unsigned int n) {
    n &= 0x09249249;
    n = (n ^ (n >> 2)) & 0x030c30c3;
    n = (n ^ (n >> 4)) & 0x0300f00f;
    n = (n ^ (n >> 8)) & 0xff0000ff;
    n = (n ^ (n >> 16)) & 0x000003ff;
    return n;
}

__device__
unsigned int interleave3(unsigned int x, unsigned int y, unsigned int z) {
    return part1by2(x) | (part1by2(y) << 1) | (part1by2(z) << 2);
}

__device__
void deinterleave3(unsigned int n, unsigned int x, unsigned int y, unsigned int z) {
    x = unpart1by2(n);
    y = unpart1by2(n >> 1);
    z = unpart1by2(n >> 2);
}


__device__
bool contains(unsigned int arr[27], int size, unsigned int member) {
    bool ret = false;

    for (int i = 0; i < size; i++)
    {
        if (arr[i] == member)
            ret = true;
    }
    return ret;
}
__device__
int getAdj(float4 pos, float H,unsigned int ret[27]) {

    int retSize = 0;
    unsigned int offset = 43;
    unsigned int morton_x = unsigned int((pos.x / H) + offset);
    unsigned int morton_y = unsigned int((pos.y / H) + offset);
    unsigned int morton_z = unsigned int((pos.z / H) + offset);

    unsigned int morton_cell;

    if (morton_x > 0 && morton_y > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y - 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x, morton_y - 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_z > 0) {
        morton_cell = interleave3(morton_x, morton_y, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0 && morton_y > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y - 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y > 0) {
        morton_cell = interleave3(morton_x, morton_y - 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    morton_cell = interleave3(morton_x, morton_y, morton_z);
    ret[retSize] = morton_cell;
    retSize++;

    //1864184 é o numero maximo que o morton code pode devolver num cubo de -2 a 2 
    if (morton_x < 1864184 && morton_y > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y - 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x < 1864184 && morton_y > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y - 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x > 0 && morton_y < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y + 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_y < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x, morton_y + 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_y < 1864184 && morton_x > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y + 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y < 1864184) {
        morton_cell = interleave3(morton_x, morton_y + 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y + 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y + 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x > 0 && morton_y > 0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x - 1, morton_y - 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y > 0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x, morton_y - 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x - 1, morton_y, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_z < 1864184) {
        morton_cell = interleave3(morton_x, morton_y, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y >0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y - 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x > 0 && morton_y < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x - 1, morton_y + 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_y < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x, morton_y + 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y + 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    return retSize;

}

__device__
float useDefaultKernel(float4 distVector, float supportRadius) {
    
    float dist = length(distVector);
    
    if (dist > supportRadius) {

        return 0.0;
    }
    else {
        //printf("Vizinho e vai devolver -> %f \n", (315 / (64 * 3.141592653589793 * pow(supportRadius, 9.0f))) * pow(supportRadius * supportRadius - dist * dist, 3.0f));
        return (315 / (64 * 3.141592653589793 * pow(supportRadius, 9.0f))) * pow(supportRadius * supportRadius - dist * dist, 3.0f);
    }
}
__device__
unsigned int hashFunction(float4 pos, double H, int size) {
    
    int p1 = 2693;
    int p2 = 3163;
    int p3 = 4091;

    int part1 = (int((pos.x / H)) * p1);
    int part2 = (int((pos.y / H)) * p2);
    int part3 = (int((pos.z / H)) * p3);

    unsigned int ret = unsigned int((part1 ^ part2 ^ part3) % size);

    return ret;
}
//----------------


__global__
void count(int * indexes , int* CellStart, int* CellEnd)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	int indexCount = indexes[i];
	atomicAdd(CellStart + indexCount, 1);

	atomicAdd(CellEnd + indexCount, 1);
	
}

void kernelWraper(int * dptrssboIndex, int* dptrssboCellStart, int* dptrssboCellEnd) {

	count<<<6750, 32 >>>(dptrssboIndex, dptrssboCellStart, dptrssboCellEnd);
	
	

	thrust::device_ptr<int> cellstartThrust = thrust::device_pointer_cast((dptrssboCellStart));
	thrust::exclusive_scan(cellstartThrust, cellstartThrust + 2000000, cellstartThrust);
	
}



__global__
void densityPressureKernel(float4 * dptrssboPosition,int* dptrssboIndex, int* dptrssboCellStart, int* dptrssboCellEnd, float* dptrssboDensity, float* dptrssboPressure, int* dptrssboAdj)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    int conta = 0;

    unsigned int ret[27];
    int retSize = 0;
    float H = 0.0457;
    unsigned int offset = 43;

    float4 pos = dptrssboPosition[index];

    unsigned int morton_x = unsigned int((pos.x / H) + offset);
    unsigned int morton_y = unsigned int((pos.y / H) + offset);
    unsigned int morton_z = unsigned int((pos.z / H) + offset);
    retSize = getAdj(pos, H, ret);

    //printf("Chegou aqui 1\n");
    
    // compute density
    float sum = 1;

    //mudar isto para um var local
    dptrssboAdj[index * 500] = 0;

    for (int j = 0; j < retSize; j++)
    {
        //uint bucket = uint(adjMat[(28*b)+(j+1)]) ;
        unsigned int cell = ret[j];

        int from = dptrssboCellStart[cell];
        int to = from + dptrssboCellEnd[cell];
        //printf("Chegou aqui 3\n");
        for (int i = from; i < to; i++)
        {


            float4 arg;
            arg = pos - dptrssboPosition[i];

            float ker_res = useDefaultKernel(arg, H);

            sum += ker_res;
            //printf("Chegou aqui 4\n");

            if (ker_res != 0) {

                int vizinhos = dptrssboAdj[index * 500];
                //Aqui é MAXADJ porque no maximo vai dar para guardar 99 vizinhos mais 1 posição para dizer quantos vizinhos se guardou
                dptrssboAdj[index * 500 + vizinhos + 1] = i;
                dptrssboAdj[index * 500]++;
                //printf("Chegou aqui 5\n");
            }


        }
    }

    //printf("Chegou aqui 6 vizinhos %d naovizinhos %d\n",vizinhos,naovizinhos);
    dptrssboDensity[index] = sum *0.02;

    //printf("Chegou aqui 7\n");
    // compute pressure
    dptrssboPressure[index] = 3.0 * ((sum * 0.02) - 998.29);
}

__device__
float useDefaultKernel_laplacian(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius)
        return 0.0f;
    else
        return -(945 / (32 * 3.141592653589793 * pow(supportRadius, 9.0f))) * (supportRadius * supportRadius - dist * dist) * (3 * supportRadius * supportRadius - 7 * dist * dist);
}
__device__
float useViscosityKernel_laplacian(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius)
        return 0.0f;
    else
        return (45 / (3.141592653589793 * pow(supportRadius, 6.0f))) * (supportRadius - dist);
}
__device__
float4 useDefaultKernel_gradient(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius) {
        return make_float4(0);
    }
    else {
        return -(distVector * (945 / (32 * 3.141592653589793 * pow(supportRadius, 9.0f))) * pow(supportRadius * supportRadius - dist * dist, 2.0f));
    }
}



__device__
float4 usePressureKernel_gradient(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius) {

        return make_float4(0);

    }
    else
    {

        float4 normalized;
        normalized = normalize(distVector);

        return -(normalized * (45 / (3.141592653589793 * pow(supportRadius, 6.0f))) * pow(supportRadius - dist, 2.0f));

    }
}

__global__
void ForceKernel(float4* dptrssboPosition, float4* dptrssboVelocity, float4* dptrssboForce, float* dptrssboDensity, float* dptrssboPressure, float4* dptrssboGravity, float4* dptrssboSurfaceNormal, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, int* dptrssboAdj)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ret[27];
    int retSize = 0;
    float H = 0.0457;

    float4 pos = dptrssboPosition[index];
    float des = dptrssboDensity[index];

    retSize = getAdj(pos, H, ret);

    float4 sumViscosity = make_float4(0);

    float4 sumForce = make_float4(0);


    float4 sumSurfaceNormal = make_float4(0);

    float sum = 0;
    float MASS = 0.02;

    int count;
    count = 0;

    int vizinhos = dptrssboAdj[index * 500];

    for (int i = 0; i < vizinhos; i++) {
        //Viscosity
        int iAdj = dptrssboAdj[index * 500 + i + 1];
        float4 posAdj = dptrssboPosition[iAdj];
        float densityAdj = dptrssboDensity[iAdj];
        if (pos.x == posAdj.x && pos.y == posAdj.y && pos.z == posAdj.z)
            continue;
        float4 arg;
        arg = pos - posAdj;

        sumViscosity += useViscosityKernel_laplacian(arg, H) * (dptrssboVelocity[iAdj] - dptrssboVelocity[index]) * (1 / dptrssboDensity[iAdj]);

        //Force

        arg = pos - posAdj;


        //printf("ret Kernel %f %f %f \n", usePressureKernel_gradient(arg, H).x, usePressureKernel_gradient(arg, H).y, usePressureKernel_gradient(arg, H).z);
        sumForce += usePressureKernel_gradient(arg, H) * (dptrssboPressure[index] / (des * des) + dptrssboPressure[iAdj] / (densityAdj * densityAdj));

        //SurfaceNormal
        arg = pos - posAdj;

        sumSurfaceNormal += useDefaultKernel_gradient(arg, H) * (1 / densityAdj);
    }

    dptrssboViscosity[index] = sumViscosity * MASS * 3.5;

    dptrssboForce[index] = -(sumForce * MASS * des);
    //debug[index] = vec4(density[index]);

    //tempPosition[index]=vec4(sumForce[0],sumForce[1],sumForce[2 ],0);

    dptrssboSurfaceNormal[index] = sumSurfaceNormal * MASS;

    float4 sn = sumSurfaceNormal * MASS;

    dptrssboGravity[index] = make_float4(0.0, -9.8, 0.0, 0.0) * des;



    if (length(sn) >= 7.065) {

        for (int i = 0; i < vizinhos; i++) {
            int iAdj = dptrssboAdj[index * 500 + i + 1];
            float4 posAdj = dptrssboPosition[iAdj];
            if (pos.x == posAdj.x && pos.y == posAdj.y && pos.z == posAdj.z)
                continue;
            float4 arg;
            arg = pos - posAdj;


            sum += (MASS / dptrssboDensity[iAdj]) * useDefaultKernel_laplacian(arg, H);
        }




        float4 surfaceNormalNormalized;

        surfaceNormalNormalized = normalize(sn);

        dptrssboSurfaceTension[index] = -(surfaceNormalNormalized * 0.0728 * sum);

    }
    else {
        dptrssboSurfaceTension[index] = make_float4(0);
    }
}

__device__
bool detectCollision(float4 pos,float * contactx, float* contacty, float* contactz, float* normalx, float* normaly, float* normalz,int index) {
    
    float4 contactPoint = make_float4(0);
    float4 unitSurfaceNormal = make_float4(0);
    float XMIN = -2;
    float YMIN = -2;
    float ZMIN = -2;

    float XMAX = 2;
    float YMAX = 2;
    float ZMAX = 2;

    float DECLIVE = 0.0;

    float newx = pos.x + XMAX;
    float temp = (XMAX + XMAX) - newx;
    temp = temp / (XMAX + XMAX); //devolve 1 quando newx é 0, ou seja, a particula esta encostada a parede esquerda
                                // devolve 0 quando esta encostada a parede direita

    float newy = YMIN + (temp * DECLIVE);

    if (pos.x <= XMAX && pos.x >= XMIN && pos.y <= YMAX && pos.y >= newy && pos.z <= ZMAX && pos.z >= ZMIN)
        return false;

    int maxComponent = 0;
    float maxValue = abs(pos.x);
    //Por causa do declive temos de ter isso em conta ao encontrar o maxvalue. (se nao fizer + temp*declive as vezes da como max component o Z quando na realidade deveria ter sido o Y, so nao foi por causa do declive)
    if (maxValue < abs(pos.y) + (temp * DECLIVE)) {
        maxComponent = 1;
        maxValue = abs(pos.y) + (temp * DECLIVE);
    }
    if (maxValue < abs(pos.z)) {
        maxComponent = 2;
        maxValue = abs(pos.z);
    }
    // 'unitSurfaceNormal' is based on the current position component with the largest absolute value

    
    switch (maxComponent) {
    case 0:
        if (pos.x < XMIN) {
            contactPoint = make_float4(XMIN, pos.y, pos.z, 0);

            if (pos.y < newy)     contactPoint.y = newy;
            else if (pos.y > YMAX) contactPoint.y = YMAX;
            if (pos.z < ZMIN)     contactPoint.z = ZMIN;
            else if (pos.z > ZMAX) contactPoint.z = ZMAX;

            unitSurfaceNormal = make_float4(1, 0, 0, 0);


        }
        else if (pos.x > XMAX) {
            contactPoint = make_float4(XMAX, pos.y, pos.z, 0);

            if (pos.y < newy)     contactPoint.y = newy;
            else if (pos.y > YMAX) contactPoint.y = YMAX;
            if (pos.z < ZMIN)     contactPoint.z = ZMIN;
            else if (pos.z > ZMAX) contactPoint.z = ZMAX;

            unitSurfaceNormal = make_float4(-1, 0, 0, 0);


        }
        break;
    case 1:
        
        if (pos.y < newy) {
            contactPoint = make_float4(pos.x, newy, pos.z, 0);

            if (pos.x < XMIN)     contactPoint.x = XMIN;
            else if (pos.x > XMAX) contactPoint.x = XMAX;
            if (pos.z < ZMIN)     contactPoint.z = ZMIN;
            else if (pos.z > ZMAX) contactPoint.z = ZMAX;

            //unitSurfaceNormal = vec4(DECLIVE,1-DECLIVE,0,0);
            unitSurfaceNormal = make_float4(DECLIVE, 1.0 - DECLIVE, 0, 0);

        }
        else if (pos.y > YMAX) {
            contactPoint = make_float4(pos.x, YMAX, pos.z, 0);

            if (pos.x < XMIN)     contactPoint.x = XMIN;
            else if (pos.x > XMAX) contactPoint.x = XMAX;
            if (pos.z < ZMIN)     contactPoint.z = ZMIN;
            else if (pos.z > ZMAX) contactPoint.z = ZMAX;

            unitSurfaceNormal = make_float4(0, -1, 0, 0);

        }
        break;
    case 2:
        if (pos.z < ZMIN) {

            contactPoint = make_float4(pos.x, pos.y, ZMIN, 0);

            if (pos.x < XMIN)     contactPoint.x = XMIN;
            else if (pos.x > XMAX) contactPoint.x = XMAX;
            if (pos.y < newy)     contactPoint.y = newy;
            else if (pos.y > YMAX) contactPoint.y = YMAX;
            unitSurfaceNormal = make_float4(0, 0, 1, 0);


        }
        else if (pos.z > ZMAX) {
            contactPoint = make_float4(pos.x, pos.y, ZMAX, 0);

            if (pos.x < XMIN)     contactPoint.x = XMIN;
            else if (pos.x > XMAX) contactPoint.x = XMAX;
            if (pos.y < newy)     contactPoint.y = newy;
            else if (pos.y > YMAX) contactPoint.y = YMAX;
            unitSurfaceNormal = make_float4(0, 0, -1, 0);

        }
        break;
    }

    //printf("Contact point %f %f %f\n", contactPoint.x, contactPoint.y, contactPoint.z);

    //printf("Normal %f %f %f\n", unitSurfaceNormal.x, unitSurfaceNormal.y, unitSurfaceNormal.z);

    *contactx = contactPoint.x;
    *contacty = contactPoint.y;
    *contactz = contactPoint.z;

    *normalx = unitSurfaceNormal.x;
    *normaly = unitSurfaceNormal.y;
    *normalz = unitSurfaceNormal.z;

    
    //printf("INSIDE pos %f %f %f maxvalue %f abs posy %f temp %f index %d maxComponent %d \n", pos.x, pos.y, pos.z, maxValue, abs(pos.y), temp, index,maxComponent);
    //printf("INSIDE pos %f %f %f maxvalue %f abs posy %f temp %f index %d \n", pos.x, pos.y, pos.z, maxValue, abs(pos.y), temp, index);

    return true;
}
__device__
float4 updateVelocity(float4 velocity, float4 unitSurfaceNormal, float penetrationDepth) {
    //ret = velocity - unitSurfaceNormal * (1 + RESTITUTION * penetrationDepth / (TIMESTEP * glm::length(velocity))) * glm::dot(velocity, unitSurfaceNormal);
    float RESTITUTION = 0.5;
    
    //se usar a var aqui ele vai me dar zero por alguma razao
    float4 ret = (velocity - unitSurfaceNormal * (1 + RESTITUTION * penetrationDepth / (0.01 * length(velocity))) * dot(unitSurfaceNormal, velocity));
    //printf("(TIMESTEP * length(velocity)) %f  .... length(velocity) %f .... velocity %f %f %f \n",(TIMESTEP * float (length(velocity)) ),length(velocity), velocity.x, velocity.y, velocity.z);
    return ret;
}

__global__
void IntegrateKernel(float4* dptrssboPosition, float4* dptrssboVelocity, float4* dptrssboForce, float* dptrssboDensity, float4* dptrssboGravity, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, float4* dptrssboAcceleration)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float4 totalForce = make_float4(0);

    totalForce = dptrssboForce[index] + dptrssboViscosity[index] + dptrssboGravity[index] + dptrssboSurfaceTension[index];



    //employEulerIntegrator
    dptrssboAcceleration[index] = totalForce / dptrssboDensity[index];
    float TIMESTEP = 0.01;
    
    dptrssboVelocity[index] = dptrssboVelocity[index] + dptrssboAcceleration[index] * TIMESTEP;

    
    dptrssboPosition[index] = dptrssboPosition[index] + dptrssboVelocity[index] * TIMESTEP;

    

    float4 contactPoint = make_float4(0);
    float4 unitSurfaceNormal = make_float4(0);

    float cx, cy, cz, nx, ny, nz;
    
    bool retcolision = detectCollision(dptrssboPosition[index],&cx, &cy, &cz, &nx, &ny, &nz,index);

    contactPoint = make_float4(cx, cy, cz, 0);
    unitSurfaceNormal = make_float4(nx, ny, nz, 0);


    if (retcolision) {
        
        //printf("Contact point %f %f %f normal %f %f %f \n", contactPoint.x, contactPoint.y, contactPoint.z, unitSurfaceNormal.x, unitSurfaceNormal.y, unitSurfaceNormal.z);
        float4 ret=make_float4(0);
        float4 arg = make_float4(0);

        arg = dptrssboPosition[index] - contactPoint;
        
        ret =updateVelocity(dptrssboVelocity[index], unitSurfaceNormal, length(arg));
        //printf("updated vel %f %f %f unitSurfaceNormal %f %f %f len %f \n", ret.x, ret.y, ret.z, unitSurfaceNormal.x, unitSurfaceNormal.y, unitSurfaceNormal.z, length(arg));

        dptrssboVelocity[index] = ret * 0.998;

        if (length(dptrssboVelocity[index]) < 0.01)
            dptrssboVelocity[index] = make_float4(0, 0, 0, 0);


        dptrssboPosition[index] = contactPoint;
    }
}

__global__
void UpdateKernel(float4* dptrssboPosition, int* dptrssboIndex, int* dptrssboTempIndex)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    uint offset = 43;

    float H = 0.0457;

    uint morton_x = uint((dptrssboPosition[index].x / H) + offset);
    uint morton_y = uint((dptrssboPosition[index].y / H) + offset);
    uint morton_z = uint((dptrssboPosition[index].z / H) + offset);
    uint morton_cell = interleave3(morton_x, morton_y, morton_z);

    dptrssboIndex[index] = morton_cell;
    dptrssboTempIndex[index] = morton_cell;
}

__global__
void Initkernel(int* dptrssboIndex, int* dptrssboTempIndex, float4* dptrssboPosition)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    float MASS = 0.02;
    float RESTDENSITY = 998.00;

    float fluidVolume = 1000 * MASS / RESTDENSITY;
    float particleDiameter = pow(fluidVolume, 1.0f / 3.0f) / 10;
    float particleRadius = particleDiameter / 2;

    int x_pontos = 0;
    int y_pontos = 0;
    int z_pontos = 0;

    int inseridos = 0;

    int conta = 0;
    int myoffset = 0;

    int PONTOS_LADO = 60;

    float H = 0.0457;

    //Cria as particulas e poe no array temporario
    //calcula o tamanho de cada bucket, tambem temporario
    for (float x = -particleRadius * PONTOS_LADO; x_pontos < PONTOS_LADO; x += particleDiameter, x_pontos++) {
        for (float y = -particleRadius * PONTOS_LADO, y_pontos = 0; y_pontos < PONTOS_LADO; y += particleDiameter, y_pontos++) {
            for (float z = -particleRadius * PONTOS_LADO, z_pontos = 0; z_pontos < PONTOS_LADO; z += particleDiameter, z_pontos++) {

                //Este offset é calculado assim: como temos os pontos -2 isso da a celula -43 (-2 / H), assim para os valores enviados para a funcao que calcula o morton code serem sempre 0 ou maior, temos de somar esses 43
                uint offset = 43;

                uint morton_x = uint((x / H) + offset);
                uint morton_y = uint((y / H) + offset);
                uint morton_z = uint((z / H) + offset);
                uint morton_cell = interleave3(morton_x, morton_y, morton_z);



                //tempPosition agora vai guardar o index do codigo morton e o position guarda a posição normal
                dptrssboIndex[inseridos] = morton_cell;
                dptrssboTempIndex[inseridos] = morton_cell;
                dptrssboPosition[inseridos] = make_float4(x, y, z, 0);
                inseridos++;


            }
        }
    }
}

void cudaDensityPressure(float4 * dptrssboPosition,int * dptrssboIndex,int * dptrssboCellStart,int * dptrssboCellEnd,float * dptrssboDensity, float * dptrssboPressure, int * dptrssboAdj) {
	densityPressureKernel << <1125, 192 >> > (dptrssboPosition,dptrssboIndex, dptrssboCellStart, dptrssboCellEnd, dptrssboDensity, dptrssboPressure, dptrssboAdj);
}

void cudaForce(float4* dptrssboPosition, float4* dptrssboVelocity, float4* dptrssboForce, float* dptrssboDensity, float* dptrssboPressure, float4* dptrssboGravity, float4* dptrssboSurfaceNormal, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, int* dptrssboAdj) {
    ForceKernel << <1125, 192 >> > (dptrssboPosition, dptrssboVelocity, dptrssboForce, dptrssboDensity, dptrssboPressure, dptrssboGravity, dptrssboSurfaceNormal, dptrssboSurfaceTension, dptrssboViscosity, dptrssboAdj);
}

void cudaIntegrate(float4 * dptrssboPosition, float4 * dptrssboVelocity, float4 * dptrssboForce,float* dptrssboDensity, float4* dptrssboGravity, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, float4* dptrssboAcceleration) {
    
    IntegrateKernel << <1125, 192 >> > (dptrssboPosition, dptrssboVelocity, dptrssboForce, dptrssboDensity, dptrssboGravity, dptrssboSurfaceTension, dptrssboViscosity, dptrssboAcceleration);
}

void cudaUpdateIndex(float4* dptrssboPosition, int* dptrssboIndex, int* dptrssboTempIndex) {

    UpdateKernel << <1125, 192 >> > (dptrssboPosition, dptrssboIndex, dptrssboTempIndex);
}

void cudaInit(int * dptrssboIndex, int* dptrssboTempIndex,float4 *  dptrssboPosition) {
    Initkernel << <1125, 192 >> > (dptrssboIndex, dptrssboTempIndex, dptrssboPosition);
}