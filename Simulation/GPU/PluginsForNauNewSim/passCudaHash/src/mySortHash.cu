#include "hip/hip_runtime.h"
#include "mySortHash.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "cutil_math.h"
#include "nau/debug/profile.h"
//#include <math_functions.h>



void mysort(int * index1, float4 * values1, int* index2, float4 * values2,int particles){
	
	
    
        thrust::device_ptr<int> i1buff = thrust::device_pointer_cast((index1));
        thrust::device_ptr<float4> v1buff = thrust::device_pointer_cast((values1));
        thrust::device_ptr<int> i2buff = thrust::device_pointer_cast((index2));
        thrust::device_ptr<float4> v2buff = thrust::device_pointer_cast((values2));
    
    {
        PROFILE("Sort_by_key_Pos");
        thrust::sort_by_key(i1buff, i1buff + particles, v1buff);
    }
    {
        PROFILE("Sort_by_key_Velocity");
        thrust::sort_by_key(i2buff, i2buff + particles, v2buff);
    }
}

//functions for density Pressure
//Morton code --------------------------------------------


__device__
unsigned int part1by2(unsigned int n) {
    n &= 0x000003ff;
    n = (n ^ (n << 16)) & 0xff0000ff;
    n = (n ^ (n << 8)) & 0x0300f00f;
    n = (n ^ (n << 4)) & 0x030c30c3;
    n = (n ^ (n << 2)) & 0x09249249;
    return n;
}

__device__
unsigned int unpart1by2(unsigned int n) {
    n &= 0x09249249;
    n = (n ^ (n >> 2)) & 0x030c30c3;
    n = (n ^ (n >> 4)) & 0x0300f00f;
    n = (n ^ (n >> 8)) & 0xff0000ff;
    n = (n ^ (n >> 16)) & 0x000003ff;
    return n;
}

__device__
unsigned int interleave3(unsigned int x, unsigned int y, unsigned int z) {
    return part1by2(x) | (part1by2(y) << 1) | (part1by2(z) << 2);
}

__device__
void deinterleave3(unsigned int n, unsigned int x, unsigned int y, unsigned int z) {
    x = unpart1by2(n);
    y = unpart1by2(n >> 1);
    z = unpart1by2(n >> 2);
}


__device__
bool contains(unsigned int arr[27], int size, unsigned int member) {
    bool ret = false;

    for (int i = 0; i < size; i++)
    {
        if (arr[i] == member)
            ret = true;
    }
    return ret;
}
__device__
int getAdj(float4 pos, float H,unsigned int ret[27]) {

    int retSize = 0;
    unsigned int offset = 43;
    unsigned int morton_x = unsigned int((pos.x / H) + offset);
    unsigned int morton_y = unsigned int((pos.y / H) + offset);
    unsigned int morton_z = unsigned int((pos.z / H) + offset);

    unsigned int morton_cell;

    if (morton_x > 0 && morton_y > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y - 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x, morton_y - 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_z > 0) {
        morton_cell = interleave3(morton_x, morton_y, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0 && morton_y > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y - 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y > 0) {
        morton_cell = interleave3(morton_x, morton_y - 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    morton_cell = interleave3(morton_x, morton_y, morton_z);
    ret[retSize] = morton_cell;
    retSize++;

    //1864184 � o numero maximo que o morton code pode devolver num cubo de -2 a 2 
    if (morton_x < 1864184 && morton_y > 0 && morton_z > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y - 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x < 1864184 && morton_y > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y - 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x > 0 && morton_y < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y + 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_y < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x, morton_y + 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_y < 1864184 && morton_x > 0) {
        morton_cell = interleave3(morton_x - 1, morton_y + 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y < 1864184) {
        morton_cell = interleave3(morton_x, morton_y + 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y < 1864184 && morton_z > 0) {
        morton_cell = interleave3(morton_x + 1, morton_y + 1, morton_z - 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y + 1, morton_z);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x > 0 && morton_y > 0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x - 1, morton_y - 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_y > 0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x, morton_y - 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x > 0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x - 1, morton_y, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_z < 1864184) {
        morton_cell = interleave3(morton_x, morton_y, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y >0 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y - 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }
    if (morton_x < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x > 0 && morton_y < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x - 1, morton_y + 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_y < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x, morton_y + 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    if (morton_x < 1864184 && morton_y < 1864184 && morton_z < 1864184) {
        morton_cell = interleave3(morton_x + 1, morton_y + 1, morton_z + 1);
        ret[retSize] = morton_cell;
        retSize++;
    }

    return retSize;

}

__device__
float useDefaultKernel(float4 distVector, float supportRadius) {
    
    float dist = length(distVector);
    
    if (dist > supportRadius) {

        return 0.0;
    }
    else {
        //printf("Vizinho e vai devolver -> %f \n", (315 / (64 * 3.141592653589793 * pow(supportRadius, 9.0f))) * pow(supportRadius * supportRadius - dist * dist, 3.0f));
        return (315 / (64 * 3.141592653589793 * pow(supportRadius, 9.0f))) * pow(supportRadius * supportRadius - dist * dist, 3.0f);
    }
}
__device__
unsigned int hashFunction(float4 pos, double H, int size) {
    
    int p1 = 2693;
    int p2 = 3163;
    int p3 = 4091;

    int part1 = (int((pos.x / H)) * p1);
    int part2 = (int((pos.y / H)) * p2);
    int part3 = (int((pos.z / H)) * p3);

    unsigned int ret = unsigned int((part1 ^ part2 ^ part3) % size);

    return ret;
}
//----------------


__global__
void count(int * indexes , int* CellStart, int* CellEnd)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	int indexCount = indexes[i];
    
	atomicAdd(CellStart + indexCount, 1);

    

	atomicAdd(CellEnd + indexCount, 1);
	
}

void kernelWraper(int * dptrssboIndex, int* dptrssboCellStart, int* dptrssboCellEnd,int nrParticles,int hashSize) {

    //Alterar aqui quando se muda o tamanho das particulas

    int x = nrParticles / 64;

	count<<<x, 64 >>>(dptrssboIndex, dptrssboCellStart, dptrssboCellEnd);

	
	thrust::device_ptr<int> cellstartThrust = thrust::device_pointer_cast((dptrssboCellStart));
	thrust::exclusive_scan(cellstartThrust, cellstartThrust + hashSize, cellstartThrust);
}



__global__
void densityPressureKernel(float4 * dptrssboPosition,int* dptrssboIndex, int* dptrssboCellStart, int* dptrssboCellEnd, float* dptrssboDensity, float* dptrssboPressure, int* dptrssboAdj)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    int conta = 0;

    unsigned int ret[27];
    int retSize = 0;
    float H = 0.0457;
    unsigned int offset = 43;
    unsigned int morton_x = unsigned int((dptrssboPosition[index].x / H) + offset);
    unsigned int morton_y = unsigned int((dptrssboPosition[index].y / H) + offset);
    unsigned int morton_z = unsigned int((dptrssboPosition[index].z / H) + offset);
    retSize = getAdj(dptrssboPosition[index], H, ret);

    //printf("Chegou aqui 1\n");
    
    // compute density
    float sum = 1;

    //mudar isto para um var local
    dptrssboAdj[index * 500] = 0;
    //printf("Chegou aqui 2\n");
    int vizinhos = 0;

    int naovizinhos = 0;

    for (int j = 0; j < retSize; j++)
    {
        //uint bucket = uint(adjMat[(28*b)+(j+1)]) ;
        unsigned int cell = ret[j];

        int from = dptrssboCellStart[cell];
        int to = from + dptrssboCellEnd[cell];
        //printf("Chegou aqui 3\n");
        for (int i = from; i < to; i++)
        {


            float4 arg;
            arg = dptrssboPosition[index] - dptrssboPosition[i];

            float ker_res = useDefaultKernel(arg, H);

            sum += 0.02 * ker_res;
            //printf("Chegou aqui 4\n");

            if (ker_res != 0) {

                conta++;
                int vizinhos = dptrssboAdj[index * 500];
                //Aqui � MAXADJ porque no maximo vai dar para guardar 99 vizinhos mais 1 posi��o para dizer quantos vizinhos se guardou
                dptrssboAdj[index * 500 + vizinhos + 1] = i;
                dptrssboAdj[index * 500]++;
                vizinhos++;
                //printf("Chegou aqui 5\n");
            }
            else
                naovizinhos++;


        }
    }

    //printf("Chegou aqui 6 vizinhos %d naovizinhos %d\n",vizinhos,naovizinhos);
    dptrssboDensity[index] = sum;

    //printf("Chegou aqui 7\n");
    // compute pressure
    dptrssboPressure[index] = 3.0 * (sum - 998.29);
}

__device__
float useDefaultKernel_laplacian(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius)
        return 0.0f;
    else
        return -(945 / (32 * 3.141592653589793 * pow(supportRadius, 9.0f))) * (supportRadius * supportRadius - dist * dist) * (3 * supportRadius * supportRadius - 7 * dist * dist);
}
__device__
float useViscosityKernel_laplacian(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius)
        return 0.0f;
    else
        return (45 / (3.141592653589793 * pow(supportRadius, 6.0f))) * (supportRadius - dist);
}
__device__
float4 useDefaultKernel_gradient(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius) {
        return make_float4(0);
    }
    else {
        return -(distVector * (945 / (32 * 3.141592653589793 * pow(supportRadius, 9.0f))) * pow(supportRadius * supportRadius - dist * dist, 2.0f));
    }
}



__device__
float4 usePressureKernel_gradient(float4 distVector, float supportRadius) {
    float dist = length(distVector);
    if (dist > supportRadius) {

        return make_float4(0);

    }
    else
    {

        float4 normalized;
        normalized = normalize(distVector);

        return -(normalized * (45 / (3.141592653589793 * pow(supportRadius, 6.0f))) * pow(supportRadius - dist, 2.0f));

    }
}

__global__
void ForceKernel(float4* dptrssboPosition, float4* dptrssboVelocity, float4* dptrssboForce, float* dptrssboDensity, float* dptrssboPressure, float4* dptrssboGravity, float4* dptrssboSurfaceNormal, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, int* dptrssboAdj)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ret[27];
    int retSize = 0;
    float H = 0.0457;

    retSize = getAdj(dptrssboPosition[index], H, ret);

    float4 sumViscosity = make_float4(0);

    float4 sumForce= make_float4(0);
    

    float4 sumSurfaceNormal= make_float4(0);
    
    float sum = 0;
    float MASS = 0.02;

    int count;
    count = 0;

    int vizinhos = dptrssboAdj[index * 500];

    for (int i = 0; i < vizinhos; i++) {
        //Viscosity

        if (dptrssboPosition[index].x == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].x && dptrssboPosition[index].y == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].y && dptrssboPosition[index].z == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].z)
            continue;
        float4 arg;
        arg = dptrssboPosition[index] - dptrssboPosition[dptrssboAdj[index * 500 + i + 1]];

        sumViscosity += useViscosityKernel_laplacian(arg, H) * (dptrssboVelocity[dptrssboAdj[index * 500 + i + 1]] - dptrssboVelocity[index]) * (MASS / dptrssboDensity[dptrssboAdj[index * 500 + i + 1]]);

        //Force
  
        arg = dptrssboPosition[index] - dptrssboPosition[dptrssboAdj[index * 500 + i + 1]];

        
        //printf("ret Kernel %f %f %f \n", usePressureKernel_gradient(arg, H).x, usePressureKernel_gradient(arg, H).y, usePressureKernel_gradient(arg, H).z);
        sumForce += usePressureKernel_gradient(arg, H) * (dptrssboPressure[index] / (dptrssboDensity[index] * dptrssboDensity[index]) + dptrssboPressure[dptrssboAdj[index * 500 + i + 1]] / (dptrssboDensity[dptrssboAdj[index * 500 + i + 1]] * dptrssboDensity[dptrssboAdj[index * 500 + i + 1]])) * MASS;

        //SurfaceNormal
        arg = dptrssboPosition[index] - dptrssboPosition[dptrssboAdj[index * 500 + i + 1]];
        
        sumSurfaceNormal += useDefaultKernel_gradient(arg, H) * (MASS / dptrssboDensity[dptrssboAdj[index * 500 + i + 1]]);
    }

    dptrssboViscosity[index] = sumViscosity * 3.5;

    dptrssboForce[index] = -(sumForce * dptrssboDensity[index]);
    //debug[index] = vec4(density[index]);

    //tempPosition[index]=vec4(sumForce[0],sumForce[1],sumForce[2 ],0);

    dptrssboSurfaceNormal[index] = sumSurfaceNormal;

    dptrssboGravity[index] = make_float4(0.0,-9.8,0.0,0.0) * dptrssboDensity[index];



    if (length(dptrssboSurfaceNormal[index]) >= 7.065) {

        for (int i = 0; i < vizinhos; i++) {
            if (dptrssboPosition[index].x == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].x && dptrssboPosition[index].y == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].y && dptrssboPosition[index].z == dptrssboPosition[dptrssboAdj[index * 500 + i + 1]].z)
                continue;
            float4 arg;
            arg = dptrssboPosition[index] - dptrssboPosition[dptrssboAdj[index * 500 + i + 1]];
            
            
            sum += (MASS / dptrssboDensity[dptrssboAdj[index * 500 + i + 1]]) * useDefaultKernel_laplacian(arg, H);
        }




        float4 surfaceNormalNormalized;

        surfaceNormalNormalized = normalize(dptrssboSurfaceNormal[index]);

        dptrssboSurfaceTension[index] = -(surfaceNormalNormalized * 0.0728 * sum);

    }
    else {
        dptrssboSurfaceTension[index] =make_float4(0);
    }
}

__device__
bool detectCollision(float4 pos,float * contactx, float* contacty, float* contactz, float* normalx, float* normaly, float* normalz,int index) {
    
    float4 contactPoint = make_float4(0);
    float4 unitSurfaceNormal = make_float4(0);
    float XMIN = -2;
    float YMIN = -2;
    float ZMIN = -2;

    float XMAX = 2;
    float YMAX = 2;
    float ZMAX = 2;

    float DECLIVE = 0.0;

    float newx = pos.x + XMAX;
    float temp = (XMAX + XMAX) - newx;
    temp = temp / (XMAX + XMAX); //devolve 1 quando newx � 0, ou seja, a particula esta encostada a parede esquerda
                                // devolve 0 quando esta encostada a parede direita

    float newy = YMIN + (temp * DECLIVE);

    if (pos.x <= XMAX && pos.x >= XMIN && pos.y <= YMAX && pos.y >= newy && pos.z <= ZMAX && pos.z >= ZMIN)
        return false;

    int maxComponent = 0;
    float maxValue = abs(pos.x);
    //Por causa do declive temos de ter isso em conta ao encontrar o maxvalue. (se nao fizer + temp*declive as vezes da como max component o Z quando na realidade deveria ter sido o Y, so nao foi por causa do declive)
    if (maxValue < abs(pos.y) + (temp * DECLIVE)) {
        maxComponent = 1;
        maxValue = abs(pos.y) + (temp * DECLIVE);
    }
    if (maxValue < abs(pos.z)) {
        maxComponent = 2;
        maxValue = abs(pos.z);
    }
    // 'unitSurfaceNormal' is based on the current position component with the largest absolute value

    
    switch (maxComponent) {
    case 0:
        if (pos.x < XMIN) {
            contactPoint = make_float4(XMIN, pos.y, pos.z, 0);

            if (pos.y < newy)     contactPoint.y = newy;
            else if (pos.y > YMAX) contactPoint.y = YMAX;
            if (pos.z < ZMIN)     contactPoint.z = ZMIN;
            else if (pos.z > ZMAX) contactPoint.z = ZMAX;

            unitSurfaceNormal = make_float4(1, 0, 0, 0);


        }
        else if (pos.x > XMAX) {
            contactPoint = make_float4(XMAX, pos.y, pos.z, 0);

            if (pos.y < newy)     contactPoint.y = newy;
            else if (pos.y > YMAX) contactPoint.y = YMAX;
            if (pos.z < ZMIN)     contactPoint.z = ZMIN;
            else if (pos.z > ZMAX) contactPoint.z = ZMAX;

            unitSurfaceNormal = make_float4(-1, 0, 0, 0);


        }
        break;
    case 1:
        
        if (pos.y < newy) {
            contactPoint = make_float4(pos.x, newy, pos.z, 0);

            if (pos.x < XMIN)     contactPoint.x = XMIN;
            else if (pos.x > XMAX) contactPoint.x = XMAX;
            if (pos.z < ZMIN)     contactPoint.z = ZMIN;
            else if (pos.z > ZMAX) contactPoint.z = ZMAX;

            //unitSurfaceNormal = vec4(DECLIVE,1-DECLIVE,0,0);
            unitSurfaceNormal = make_float4(DECLIVE, 1.0 - DECLIVE, 0, 0);

        }
        else if (pos.y > YMAX) {
            contactPoint = make_float4(pos.x, YMAX, pos.z, 0);

            if (pos.x < XMIN)     contactPoint.x = XMIN;
            else if (pos.x > XMAX) contactPoint.x = XMAX;
            if (pos.z < ZMIN)     contactPoint.z = ZMIN;
            else if (pos.z > ZMAX) contactPoint.z = ZMAX;

            unitSurfaceNormal = make_float4(0, -1, 0, 0);

        }
        break;
    case 2:
        if (pos.z < ZMIN) {

            contactPoint = make_float4(pos.x, pos.y, ZMIN, 0);

            if (pos.x < XMIN)     contactPoint.x = XMIN;
            else if (pos.x > XMAX) contactPoint.x = XMAX;
            if (pos.y < newy)     contactPoint.y = newy;
            else if (pos.y > YMAX) contactPoint.y = YMAX;
            unitSurfaceNormal = make_float4(0, 0, 1, 0);


        }
        else if (pos.z > ZMAX) {
            contactPoint = make_float4(pos.x, pos.y, ZMAX, 0);

            if (pos.x < XMIN)     contactPoint.x = XMIN;
            else if (pos.x > XMAX) contactPoint.x = XMAX;
            if (pos.y < newy)     contactPoint.y = newy;
            else if (pos.y > YMAX) contactPoint.y = YMAX;
            unitSurfaceNormal = make_float4(0, 0, -1, 0);

        }
        break;
    }

    //printf("Contact point %f %f %f\n", contactPoint.x, contactPoint.y, contactPoint.z);

    //printf("Normal %f %f %f\n", unitSurfaceNormal.x, unitSurfaceNormal.y, unitSurfaceNormal.z);

    *contactx = contactPoint.x;
    *contacty = contactPoint.y;
    *contactz = contactPoint.z;

    *normalx = unitSurfaceNormal.x;
    *normaly = unitSurfaceNormal.y;
    *normalz = unitSurfaceNormal.z;

    
    //printf("INSIDE pos %f %f %f maxvalue %f abs posy %f temp %f index %d maxComponent %d \n", pos.x, pos.y, pos.z, maxValue, abs(pos.y), temp, index,maxComponent);
    //printf("INSIDE pos %f %f %f maxvalue %f abs posy %f temp %f index %d \n", pos.x, pos.y, pos.z, maxValue, abs(pos.y), temp, index);

    return true;
}
__device__
float4 updateVelocity(float4 velocity, float4 unitSurfaceNormal, float penetrationDepth) {
    //ret = velocity - unitSurfaceNormal * (1 + RESTITUTION * penetrationDepth / (TIMESTEP * glm::length(velocity))) * glm::dot(velocity, unitSurfaceNormal);
    float RESTITUTION = 0.5;
    
    //se usar a var aqui ele vai me dar zero por alguma razao
    float4 ret = (velocity - unitSurfaceNormal * (1 + RESTITUTION * penetrationDepth / (0.01 * length(velocity))) * dot(unitSurfaceNormal, velocity));
    //printf("(TIMESTEP * length(velocity)) %f  .... length(velocity) %f .... velocity %f %f %f \n",(TIMESTEP * float (length(velocity)) ),length(velocity), velocity.x, velocity.y, velocity.z);
    return ret;
}

__global__
void IntegrateKernel(float4* dptrssboPosition, float4* dptrssboVelocity, float4* dptrssboForce, float* dptrssboDensity, float4* dptrssboGravity, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, float4* dptrssboAcceleration)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float4 totalForce = make_float4(0);

    totalForce = dptrssboForce[index] + dptrssboViscosity[index] + dptrssboGravity[index] + dptrssboSurfaceTension[index];



    //employEulerIntegrator
    dptrssboAcceleration[index] = totalForce / dptrssboDensity[index];
    float TIMESTEP = 0.01;
    
    dptrssboVelocity[index] = dptrssboVelocity[index] + dptrssboAcceleration[index] * TIMESTEP;

    
    dptrssboPosition[index] = dptrssboPosition[index] + dptrssboVelocity[index] * TIMESTEP;

    

    float4 contactPoint = make_float4(0);
    float4 unitSurfaceNormal = make_float4(0);

    float cx, cy, cz, nx, ny, nz;
    
    bool retcolision = detectCollision(dptrssboPosition[index],&cx, &cy, &cz, &nx, &ny, &nz,index);

    contactPoint = make_float4(cx, cy, cz, 0);
    unitSurfaceNormal = make_float4(nx, ny, nz, 0);


    if (retcolision) {
        
        //printf("Contact point %f %f %f normal %f %f %f \n", contactPoint.x, contactPoint.y, contactPoint.z, unitSurfaceNormal.x, unitSurfaceNormal.y, unitSurfaceNormal.z);
        float4 ret=make_float4(0);
        float4 arg = make_float4(0);

        arg = dptrssboPosition[index] - contactPoint;
        
        ret =updateVelocity(dptrssboVelocity[index], unitSurfaceNormal, length(arg));
        //printf("updated vel %f %f %f unitSurfaceNormal %f %f %f len %f \n", ret.x, ret.y, ret.z, unitSurfaceNormal.x, unitSurfaceNormal.y, unitSurfaceNormal.z, length(arg));

        dptrssboVelocity[index] = ret * 0.998;

        if (length(dptrssboVelocity[index]) < 0.01)
            dptrssboVelocity[index] = make_float4(0, 0, 0, 0);


        dptrssboPosition[index] = contactPoint;
    }
}

__global__
void UpdateKernel(float4* dptrssboPosition, int* dptrssboIndex, int* dptrssboTempIndex)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    uint offset = 43;

    float H = 0.0457;

    uint morton_x = uint((dptrssboPosition[index].x / H) + offset);
    uint morton_y = uint((dptrssboPosition[index].y / H) + offset);
    uint morton_z = uint((dptrssboPosition[index].z / H) + offset);
    uint morton_cell = interleave3(morton_x, morton_y, morton_z);

    dptrssboIndex[index] = morton_cell;
    dptrssboTempIndex[index] = morton_cell;
}



void cudaDensityPressure(float4 * dptrssboPosition,int * dptrssboIndex,int * dptrssboCellStart,int * dptrssboCellEnd,float * dptrssboDensity, float * dptrssboPressure, int * dptrssboAdj) {
	densityPressureKernel << <1125, 192 >> > (dptrssboPosition,dptrssboIndex, dptrssboCellStart, dptrssboCellEnd, dptrssboDensity, dptrssboPressure, dptrssboAdj);
}

void cudaForce(float4* dptrssboPosition, float4* dptrssboVelocity, float4* dptrssboForce, float* dptrssboDensity, float* dptrssboPressure, float4* dptrssboGravity, float4* dptrssboSurfaceNormal, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, int* dptrssboAdj) {
    ForceKernel << <1125, 192 >> > (dptrssboPosition, dptrssboVelocity, dptrssboForce, dptrssboDensity, dptrssboPressure, dptrssboGravity, dptrssboSurfaceNormal, dptrssboSurfaceTension, dptrssboViscosity, dptrssboAdj);
}

void cudaIntegrate(float4 * dptrssboPosition, float4 * dptrssboVelocity, float4 * dptrssboForce,float* dptrssboDensity, float4* dptrssboGravity, float4* dptrssboSurfaceTension, float4* dptrssboViscosity, float4* dptrssboAcceleration) {
    
    IntegrateKernel << <1125, 192 >> > (dptrssboPosition, dptrssboVelocity, dptrssboForce, dptrssboDensity, dptrssboGravity, dptrssboSurfaceTension, dptrssboViscosity, dptrssboAcceleration);
}

void cudaUpdateIndex(float4* dptrssboPosition, int* dptrssboIndex, int* dptrssboTempIndex) {

    UpdateKernel << <1125, 192 >> > (dptrssboPosition, dptrssboIndex, dptrssboTempIndex);
}

void cudaComputeAdjV2(int* dptrssboAdjV2) {
    //computeAdjV2Kernel << < 31250 , 64>> > (dptrssboAdjV2);
}